
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdio>
#include <iostream>
#include <algorithm>

float*
alloc_mem(size_t NX, size_t NY)
{
  float *data;
  hipMallocManaged(&data, 2 * NY * NX * sizeof(float));
  return data;
}

int
init(float* data, size_t NX, size_t NY)
{
  size_t cx = NX/2, cy = 0; /* center of ink */
  size_t rad = (NX+NY)/8; /* radius of ink */
#pragma omp parallel shared(cx, rad, cy)
  for(size_t y = 0; y < NY; y++) {
    for(size_t x = 0; x < NX; x++) {
      float v;
      if (((x-cx)*(x-cx)+(y-cy)*(y-cy)) < rad*rad) {
        v = 1.0;
      } else {
        v = 0.0;
      }
      data[y*NX+x] = v;
      data[(NY+y)*NX+x] = v;
    }
  }
  return 0;
}

int
free_mem(float* data, size_t NY)
{
  hipFree(data);
  return 0;
}

#define GPU_ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__global__ void
line_kernel(float* data, size_t from, size_t to, size_t min, size_t max, size_t NX)
{
  size_t i = min + blockIdx.x * blockDim.x + threadIdx.x;
  while (i < max) {
    //TODO CONSIDER REMOVING MODULUS (might be slow)
    if ( (i % NX != 0) && (i % NX != NX - 1) ){
      data[to+i] = 0.2 * (
          data[from+i]
          + data[from+i-1]
          + data[from+i+1]
          + data[from+i-NX]
          + data[from+i+NX]);
    }
    i +=gridDim.x*blockDim.x;
  }
}

__global__ void
block_kernel(float* data, size_t from, size_t to, size_t min, size_t max, size_t NX)
{
  size_t i = min + blockIdx.x * blockDim.x + threadIdx.x;
  while (i < max) {
    //TODO CONSIDER REMOVING MODULUS (might be slow)
    if ( (i % NX != 0) && (i % NX != NX - 1) ){
      data[to+i] = 0.2 * (
          data[from+i]
          + data[from+i-1]
          + data[from+i+1]
          + data[from+i-NX]
          + data[from+i+NX]);
    }
    i +=gridDim.x*blockDim.x;
  }
}

int
calc(float* data, size_t NX, size_t NY, int n_steps)
{
  int n_devices;
  hipGetDeviceCount(&n_devices);
  // Implement error handling
  if (n_devices < 1) return 1;
  // Prepare stream and event pointers
  hipStream_t* line_streams = (hipStream_t*)malloc(
      n_devices*sizeof(hipStream_t));
  hipStream_t* block_streams = (hipStream_t*)malloc(
      n_devices*sizeof(hipStream_t));
  hipEvent_t *done = (hipEvent_t*)malloc(
      n_devices*n_steps*sizeof(hipEvent_t));

  int last_to;

#pragma omp parallel num_threads(n_devices)
  {
    int device_n = omp_get_thread_num();
    hipSetDevice(device_n);
    // Prepare device streams and events
    hipStreamCreate(&line_streams[device_n]);
    hipStreamCreate(&block_streams[device_n]);
    if (n_devices > 1) {
      for (int t=0; t<n_steps; ++t) {
        hipEventCreate(&done[device_n*n_steps+t]);
      }
    }
    // Also allow peer-to-peer direct access
    if (device_n > 0) {
      hipDeviceEnablePeerAccess(device_n-1, 0);
    }
    if (device_n < n_devices-1) {
      hipDeviceEnablePeerAccess(device_n+1, 0);
    }

    // Calculate needed memory
    // -3 since one more line is needed for the from array
    // and the first and last line need not be computed
    size_t Y_min_d = std::max(
        device_n * (NY + n_devices - 3) / n_devices,
        (size_t)1);
    size_t Y_max_d = std::min(
        (device_n + 1) * (NY + n_devices - 3) / n_devices,
        NY-1);
    // +-1 to skip
    size_t min = Y_min_d * NX + 1;
    size_t max = (Y_max_d) * NX - 1;
    size_t Y_range = Y_max_d - Y_min_d;
#if 0
    printf("Device: %llu\tYmin: %llu/%llu\tYmax: %llu/%llu\n",
        device_n, Y_min_d, min, Y_max_d, max);
#endif
    fflush(0);
// Make sure all events and streams are created!
#pragma omp barrier
    for (int t = 0; t < n_steps; t++) {
#if 0
      printf("device %i step %d START\n", device_n, t);
      fflush(0);
#endif
      size_t from = NY*NX*(t%2);
      size_t to = NY*NX*((t+1)%2);
      // Calculate main bulk that this gpu is responsible for if last step is
      // done
      if (t > 0) {
        GPU_ERROR_CHECK(
            hipStreamWaitEvent(
              block_streams[device_n],
              done[device_n*n_steps+t-1],
              0)
            );
      }
      block_kernel<<<std::max(Y_range*NX/512, (size_t)1),512,0,block_streams[device_n]>>>(
          data,
          from,
          to,
          min+NX,
          max-NX,
          NX);
      // Calculate top row after potentially waiting for updated results
      if (t > 0 && device_n < n_devices - 1) {
        GPU_ERROR_CHECK(
            hipStreamWaitEvent(
              line_streams[device_n+1],
              done[(device_n+1)*n_steps+t-1],
              0)
            );
      }
      line_kernel<<<std::max(NX/16, (size_t)1),16,0,line_streams[device_n]>>>(
          data,
          from,
          to,
          max-NX,
          max,
          NX);
      // Calculate bottom row after potentially waiting for updated results
      if (t > 0 && device_n > 0) {
        GPU_ERROR_CHECK(
          hipStreamWaitEvent(
              line_streams[device_n-1],
              done[(device_n-1)*n_steps+t-1],
              0)
          );
      }
      line_kernel<<<std::max(NX/16, (size_t)1),16,0,line_streams[device_n]>>>(
          data,
          from,
          to,
          min,
          min+NX,
          NX);
      // The calculations critical for neighboring blocks are done, set event
      if (n_devices > 1) {
        GPU_ERROR_CHECK(
          hipEventRecord(
              done[device_n*n_steps+t],
              line_streams[device_n])
        );
      }

      if (device_n == 0 && t == n_steps - 1) last_to = to;
    }
  }
  hipDeviceSynchronize();
  // Release steams and events
  for (int i = 0; i<n_devices; ++i) {
    hipStreamDestroy(line_streams[i]);
    hipStreamDestroy(block_streams[i]);
    for (int j=0; j<n_steps; ++j) {
      hipEventDestroy(done[i*n_steps + j]);
    }
  }
  free(line_streams);
  free(block_streams);
  free(done);
#if 0
  for (size_t i=0; i<NY; ++i) {
    printf("%f\n", data[last_to + i*NX + NX/2]);
  }
#endif
  return 0;
}
