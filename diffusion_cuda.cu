
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdio>
#include <iostream>
#include <algorithm>

#define GPU_ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


float*
alloc_mem(size_t NX, size_t NY)
{
  float *data;
  hipMallocManaged(&data, 2 * NY * NX * sizeof(float));
  return data;
}

int
free_mem(float* data)
{
  hipFree(data);
  return 0;
}

__global__ void
kernel(float* data, size_t from, size_t to, size_t min, size_t max, size_t NX)
{
  size_t i = min + blockIdx.x * blockDim.x + threadIdx.x;
  while (i < max) {
    if ( (i % NX != 0) && (i % NX != NX - 1) ){
      data[to+i] = 0.2 * (
          data[from+i]
          + data[from+i-1]
          + data[from+i+1]
          + data[from+i-NX]
          + data[from+i+NX]);
    }
    i +=gridDim.x*blockDim.x;
  }
}

int
calc(float* data, size_t NX, size_t NY, int n_steps)
{
  int n_devices;
  hipGetDeviceCount(&n_devices);
  // Implement error handling
  if (n_devices < 1) return 1;
  // Prepare stream and event pointers
  hipStream_t* streams = (hipStream_t*)malloc(n_devices*sizeof(hipStream_t));
  hipEvent_t *done = (hipEvent_t*)malloc(
      n_devices*n_steps*sizeof(hipEvent_t));

  int last_to;

#pragma omp parallel num_threads(n_devices)
  {
    int device_n = omp_get_thread_num();
    hipSetDevice(device_n);
    // Prepare device streams and events
    hipStreamCreate(&streams[device_n]);
    if (n_devices > 1) {
      for (int t=0; t<n_steps; ++t) {
        hipEventCreate(&done[device_n*n_steps+t]);
      }
    }
    // Also allow peer-to-peer direct access
    if (device_n > 0) {
      hipDeviceEnablePeerAccess(device_n-1, 0);
    }
    if (device_n < n_devices-1) {
      hipDeviceEnablePeerAccess(device_n+1, 0);
    }

    // Calculate needed memory
    // -3 since one more line is needed for the from array
    // and the first and last line need not be computed
    size_t Y_min_d = std::max(
        device_n * (NY + n_devices - 3) / n_devices,
        (size_t)1);
    size_t Y_max_d = std::min(
        (device_n + 1) * (NY + n_devices - 3) / n_devices,
        NY-1);
    // +-1 to skip
    size_t min = Y_min_d * NX + 1;
    size_t max = (Y_max_d) * NX - 1;
    size_t Y_range = Y_max_d - Y_min_d;
#if 0
    printf("Device: %llu\tYmin: %llu/%llu\tYmax: %llu/%llu\n",
        device_n, Y_min_d, min, Y_max_d, max);
#endif
    fflush(0);
// Make sure all events and streams are created!
#pragma omp barrier
    for (int t = 0; t < n_steps; t++) {
#if 0
      printf("device %i step %d START\n", device_n, t);
      fflush(0);
#endif
      size_t from = NY*NX*(t%2);
      size_t to = NY*NX*((t+1)%2);
      if (t > 0 && device_n < n_devices - 1) {
        GPU_ERROR_CHECK(
            hipStreamWaitEvent(
              streams[device_n+1],
              done[(device_n+1)*n_steps+t-1],
              0)
            );
      }
      if (t > 0 && device_n > 0) {
        GPU_ERROR_CHECK(
          hipStreamWaitEvent(
              streams[device_n-1],
              done[(device_n-1)*n_steps+t-1],
              0)
          );
      }
      kernel<<<Y_range*NX/512,512,0,streams[device_n]>>>(
          data,
          from,
          to,
          min,
          max,
          NX);
      if (n_devices > 1) {
        GPU_ERROR_CHECK(
          hipEventRecord(
              done[device_n*n_steps+t],
              streams[device_n])
        );
      }
      if (device_n == 0 && t == n_steps - 1) last_to = to;
    }
  }
  hipDeviceSynchronize();
  // Release steams and events
  for (int i = 0; i<n_devices; ++i) {
    hipStreamDestroy(streams[i]);
    for (int j=0; j<n_steps; ++j) {
      hipEventDestroy(done[i*n_steps + j]);
    }
  }
  free(streams);
  free(done);
#if 0
  for (size_t i=0; i<NY; ++i) {
    printf("%f\n", data[last_to + i*NX + NX/2]);
  }
#endif
  return 0;
}
